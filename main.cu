#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10
#define TILE_WIDTH 16
#define MAX_WIDTH 1024
#define POOL_SIZE 2
// matrix tile set to 25 to make offset calculation easier
#define CONV_TILE 25

static int FLAGS_batch_size = 10000;
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS};
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS};

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32};
static int conv2dims[] = {5, 5, 32, 64};
static int fc1dims[]   = {1024, 128};
static int fc2dims[]   = {128, 10};

// GPU functions
__global__ void convLayerForwardBasicKernel(float * X, float * W, float * Y, int W_grid, int input_wid, int output_wid, int mask_wid, int numInput, int numOutput) {
	int output_num = blockIdx.y;
	int input_num = blockIdx.x;
	int h = blockIdx.z / W_grid * TILE_WIDTH + threadIdx.y;    //h tile index
	int w = blockIdx.z % W_grid * TILE_WIDTH + threadIdx.x;     // w tiles index
	if ((h < output_wid) && (w < output_wid)){
		float acc = 0.0f;
		for (int c = 0; c < numInput; c++) {          // input features
			for (int p = 0; p < mask_wid; p++) {     //index in tile  height in ouput feature
				for (int q = 0; q < mask_wid; q ++) {    //index in tile width in ouput feature
					acc += (X[((input_num * input_wid + (h + p)) * input_wid + (w + q)) * numInput + c] *
					W[((p * mask_wid + q) * numInput + c) * numOutput  + output_num]);
				}
			}
		}
		Y[((input_num * output_wid + h) * output_wid + w) * numOutput + output_num] = acc;
	}
}

// This function uses matrix multiply to calculate convolution, it transform the input matrix
// and load that to shared memory on the fly
// We also combined relu to the writeback part of kernel
__global__ void convLayerForwardMatrixKernel(float * X, float * W, float * Y, int input_wid, int output_wid, int numInput, int numOutput) {
	__shared__ float W_tile[CONV_TILE][25];
	__shared__ float X_tile[25][CONV_TILE];
	int bx = blockIdx.x; // test case number
	int by = blockIdx.y; // output feature map number (by*CONV_TILE+ty)
	int bz = blockIdx.z; // output feature map dimension (bz*CONV_TILE+tz)
	int tx=threadIdx.x;  // output feature map number
	int ty=threadIdx.y;  // output feature map dimension
	// if tx and ty are within range
	int i,j;
	float sum=0.0f;
	int w_index,x_index;
	// offset doesn't change during iteration so calculate them in advance
	int x_offset=(bx*input_wid*input_wid+((bz*CONV_TILE+ty)/output_wid+tx/5)*input_wid+((bz*CONV_TILE+ty)%output_wid)+tx%5)*numInput;
	int w_offset=ty*numInput*numOutput+by*CONV_TILE+tx;
	for (i=0;i<numInput;i++){
		// get indexes
		w_index=w_offset+i*numOutput;
		x_index=x_offset+i;
		__syncthreads();
		if ((tx+by*CONV_TILE)<numOutput)
			W_tile[tx][ty]=W[w_index];
		else
			W_tile[tx][ty]=0.0f;
		if ((ty+bz*CONV_TILE)<(output_wid*output_wid))
			X_tile[tx][ty]=X[x_index];
		else
			X_tile[tx][ty]=0.0f;
		__syncthreads();
		// matrix multiply
		for (j=0;j<CONV_TILE;j++){
			sum+=(W_tile[tx][j]*X_tile[j][ty]);
		}
	}
	__syncthreads();
		// output format is the same as input, not the expanded matrix format
	if (((tx+by*CONV_TILE)<numOutput)&&(((ty+bz*CONV_TILE)/output_wid)<output_wid)){
		int y_offset=(bx*output_wid*output_wid+bz*CONV_TILE+ty)*numOutput+(tx+by*CONV_TILE);
		if (sum<0.0)
			sum=0.0f;
		Y[y_offset]=sum;
	}
}

__global__ void averagePool (float * X, float * Y, int W_grid, int input_wid, int output_wid, int numInput) {
	int output_num = blockIdx.y;
	int input_num = blockIdx.x;
	int h = blockIdx.z / W_grid * TILE_WIDTH + threadIdx.y;    //h tile index
	int w = blockIdx.z % W_grid * TILE_WIDTH + threadIdx.x;     // w tiles index
	if ((h < output_wid) && (w < output_wid)){
		float sum=0.0f;
		int yoffset = ((input_num * output_wid + h) * output_wid + w) * numInput + output_num;
		for (int p = 0; p < POOL_SIZE; p++) {     //index in tile  height   in ouput feature
			for (int q = 0; q < POOL_SIZE; q ++) {    //index in tile width in ouput feature
	      sum += X[((input_num * input_wid + POOL_SIZE * h + p)* input_wid + POOL_SIZE * w + q) * numInput + output_num];
			}
		}
		Y[yoffset]=sum/4.0;
	}
}

__global__ void gpu_relu4 (float * X, int total) {
	int X_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (X_idx < total){
		if (X[X_idx]<0.0)
			X[X_idx]=0.0f;
		// X[X_idx] = (X[X_idx] < 0) ? 0 : X[X_idx];
	}
}

// NN using shared memory
__global__ void gpu_fully_forward(float *X, float *W, float *Y, int output_size, int input_size){
	__shared__ float datain[1024];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	float tmp;
	// load data in
	// eliminate relu2 by checking if input is less than 0
	if (tx<input_size)
		tmp=X[bx*blockDim.x+tx];
		if (tmp<0)
			tmp=0.0f;
		datain[tx]=tmp;
	__syncthreads();
	// calculate result
	if (tx<output_size){
		int i;
		float sum=0.0f;
		for (i=0;i<input_size;i++)
			sum+=datain[i]*W[i*output_size+tx];
		Y[bx*output_size+tx]=sum;
	}
}



static int loadData(float *x, float *y) {
  // Open the data file
  const auto file_id =
      H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset x and y
  const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT);
  const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT);

  // Get the dataset x dimensions
  const auto xspace = H5Dget_space(x_id);
  const auto xndims = H5Sget_simple_extent_ndims(xspace);
  assert(xndims == 4);

  hsize_t input_dims[xndims];
  H5Sget_simple_extent_dims(xspace, input_dims, NULL);
  if (input_dims[0] != FLAGS_batch_size) {
    std::cout << "data size does not match batch size specified!\n";
    return 1; // return error
  }
  std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
            << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

  // Read the dataset x and y
  check_success(
      H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
  check_success(
      H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

  // Close the dataset x and y
  check_success(H5Dclose(x_id));
  check_success(H5Dclose(y_id));

  // Close the file
  check_success(H5Fclose(file_id));

  // return success
  return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
  // Open the model file
  const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset
  const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT);
  const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
  const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
  const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

  // Read the dataset
  check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv1));
  check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv2));
  check_success(
      H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
  check_success(
      H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

  // Close the dataset x and y
  check_success(H5Dclose(conv1_id));
  check_success(H5Dclose(conv2_id));
  check_success(H5Dclose(fc1_id));
  check_success(H5Dclose(fc2_id));

  // Close the file
  check_success(H5Fclose(file_id));
}

// From book chapter Figure 16.4
static void conv_forward_valid(const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4]) {
  const auto filter_h   = wdims[0];
  const auto filter_w   = wdims[1];
  const auto in_channel = wdims[2];

  for (const auto i : range(0, ydims[0])) { //number of input feature maps
    for (const auto m : range(0, ydims[3])) { // number of output feature maps
      for (const auto h : range(0, ydims[1])) { // image width
        for (const auto w : range(0, ydims[2])) { // image height
          for (const auto p : range(0, filter_h)) { // filter height
            for (const auto q : range(0, filter_w)) { // filter width
              for (const auto c : range(0, in_channel)) { // number of filters
                const auto yoffset =
                    ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
                const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
                                     (h + p) * xdims[2] * xdims[3] +
                                     (w + q) * xdims[3] + c;
                const auto woffset = p * wdims[1] * wdims[2] * wdims[3] +
                                     q * wdims[2] * wdims[3] + c * wdims[3] + m;
                Y[yoffset] += X[xoffset] * W[woffset];
              }
            }
          }
        }
      }
    }
  }
}

// Recified linear unit 4d
static void relu4(float *X, const int xdims[4]) {
  for (const auto i : range(0, xdims[0] * xdims[1] * xdims[2] * xdims[3])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

// Recified linear unit 2d
static void relu2(float *X, const int xdims[2]) {
  for (const auto i : range(0, xdims[0] * xdims[1])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

// From book chapter Figure 16.5
static void average_pool(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4]) {
  for (const auto i : range(0, ydims[0])) {
    for (const auto m : range(0, ydims[3])) {
      for (const auto w : range(0, ydims[2])) {
        for (const auto h : range(0, ydims[1])) {
          for (const auto p : range(0, pool_size)) {
            for (const auto q : range(0, pool_size)) {
              const auto yoffset =
                  ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
              const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
                                   (pool_size * h + p) * xdims[2] * xdims[3] +
                                   (pool_size * w + q) * xdims[3] + m;
              Y[yoffset] += X[xoffset] / (1.0f * pool_size * pool_size);
            }
          }
        }
      }
    }
  }
}

static void fully_forward(const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2]) {
  for (const auto i : range(0, xdims[0])) {
    for (const auto j : range(0, wdims[1])) {
      float sum = 0;
      for (const auto k : range(0, xdims[1])) {
        sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
      }
      Y[i * wdims[1] + j] = sum;
    }
  }
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) {
  for (const auto i : range(0, xdims[0])) {
    auto max_idx = 0;
    auto max     = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
  auto a = zeros<float>(adims);
  conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);

	// int i,j,k,l;
  /// relu layer
  relu4(a, adims);

  // average pooling
  const int pool_size = 2;
  const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                       adims[3]};
  auto b = zeros<float>(bdims);
  average_pool(a, adims, pool_size, b, bdims);
	//
	// for (i=0;i<10;i++){
	// 	for (j=0;j<24;j++){
	// 		for (k=0;k<24;k++){
	// 			for (l=0;l<32;l++)
	// 				printf("%.4f ",a[i*32*24*24+j*32*24+k*32+l]);
	// 				printf("\n");
	// 		}
	// 		printf("\n");
	// 	}
	// 	printf("\n\n");
	// }

  // conv layer
  const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  auto c = zeros<float>(cdims);
  conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);

  // relu
  relu4(c, cdims);

  // average pooling
  const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  auto d = zeros<float>(ddims);
  average_pool(c, cdims, pool_size, d, ddims);


  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  auto e            = zeros<float>(edims);
  fully_forward(d, ddims2, fc1, fc1dims, e, edims);

  // relu
  relu2(e, edims);

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};
  auto f            = zeros<float>(fdims);
  fully_forward(e, edims, fc2, fc2dims, f, fdims);
	// for (i=0;i<10;i++){
	// 	for (j=0;j<10;j++){
	// 		printf("%.4f ",f[i*10+j]);
	// 	}
	// 	printf("\n");
	// }

  argmax(f, fdims, out);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation_gpu(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  float *conv1_input;
  float *conv1_output;
  float *conv2_input;
  float *conv2_output;
  float *W1;
  float *W2;
  float *NN_L1_input;
	float *NN_L2_input;
	float *NN_output_gpu;
	float *NN_L1_weights;
	float *NN_L2_weights;
	int argdim[2]={xdims[0],fc2dims[1]};
	float *argmax_input=zeros<float>(argdim);
  int x1dim[4]={xdims[0],xdims[1],xdims[2],xdims[3]};
  int y1dim[4]={xdims[0],xdims[1]-conv1dims[0]+1,xdims[2]-conv1dims[1]+1,conv1dims[3]};
  int x2dim[4]={xdims[0],y1dim[1]/2,y1dim[2]/2,y1dim[3]};
  int y2dim[4]={xdims[0],x2dim[1]-conv2dims[0]+1,x2dim[2]-conv2dims[1]+1,conv2dims[3]};
  int NN_1_dim[4]={xdims[0],y2dim[1]/2,y2dim[2]/2,y2dim[3]};
	int NN_2_dim[2]={xdims[0],fc1dims[1]};
	// allocate global memory
  check_success(hipMalloc(&conv1_input,sizeof(float)*x1dim[0]*x1dim[1]*x1dim[2]*x1dim[3]));
  check_success(hipMalloc(&conv1_output,sizeof(float)*y1dim[0]*y1dim[1]*y1dim[2]*y1dim[3]));
  check_success(hipMalloc(&conv2_input,sizeof(float)*x2dim[0]*x2dim[1]*x2dim[2]*x2dim[3]));
  check_success(hipMalloc(&conv2_output,sizeof(float)*y2dim[0]*y2dim[1]*y2dim[2]*y2dim[3]));
  check_success(hipMalloc(&W1,sizeof(float)*conv1dims[0]*conv1dims[1]*conv1dims[2]*conv1dims[3]));
  check_success(hipMalloc(&W2,sizeof(float)*conv2dims[0]*conv2dims[1]*conv2dims[2]*conv2dims[3]));
  check_success(hipMalloc(&NN_L1_input,sizeof(float)*NN_1_dim[0]*NN_1_dim[1]*NN_1_dim[2]*NN_1_dim[3]));
  check_success(hipMalloc(&NN_L2_input,sizeof(float)*NN_2_dim[0]*NN_2_dim[1]));
  check_success(hipMalloc(&NN_L1_weights,sizeof(float)*fc1dims[0]*fc1dims[1]));
  check_success(hipMalloc(&NN_L2_weights,sizeof(float)*fc2dims[0]*fc2dims[1]));
  check_success(hipMalloc(&NN_output_gpu,sizeof(float)*argdim[0]*argdim[1]));
  check_success(hipMemcpy(conv1_input,x,sizeof(float)*x1dim[0]*x1dim[1]*x1dim[2]*x1dim[3],hipMemcpyHostToDevice));
  check_success(hipMemcpy(W2,conv2,sizeof(float)*conv2dims[0]*conv2dims[1]*conv2dims[2]*conv2dims[3],hipMemcpyHostToDevice));
	check_success(hipMemcpy(W1,conv1,sizeof(float)*conv1dims[0]*conv1dims[1]*conv1dims[2]*conv1dims[3],hipMemcpyHostToDevice));
	check_success(hipMemcpy(NN_L1_weights,fc1,sizeof(float)*fc1dims[0]*fc1dims[1],hipMemcpyHostToDevice));
	check_success(hipMemcpy(NN_L2_weights,fc2,sizeof(float)*fc2dims[0]*fc2dims[1],hipMemcpyHostToDevice));
  int Y=y1dim[1]*y1dim[2]/CONV_TILE;
	if ((y1dim[1]*y1dim[2])%CONV_TILE)
		Y++;
	int X=y1dim[3]/CONV_TILE;
	if (y1dim[3]%CONV_TILE)
		X++;
	int W_grid=0;
  dim3 conv1_block (CONV_TILE, CONV_TILE,1);
  dim3 conv1_grid (xdims[0],X,Y);
  convLayerForwardMatrixKernel<<<conv1_grid,conv1_block>>>(conv1_input,W1,conv1_output,x1dim[1],y1dim[1],x1dim[3],y1dim[3]);
  // average pool
  W_grid = x2dim[1] / TILE_WIDTH;
  int H_grid = x2dim[2] / TILE_WIDTH;
  if (y1dim[1]%TILE_WIDTH){
    W_grid++;
    H_grid++;
  }
  Y = H_grid * W_grid;
  dim3 avg_pool_1_block (TILE_WIDTH, TILE_WIDTH,1);
  dim3 avg_pool_1_grid (xdims[0], x2dim[3], Y);
  averagePool<<<avg_pool_1_grid,avg_pool_1_block>>>(conv1_output,conv2_input,W_grid,y1dim[1],x2dim[1],y1dim[3]);
	// second conv
  Y=y2dim[1]*y2dim[2]/CONV_TILE;
	if ((y2dim[1]*y2dim[2])%CONV_TILE)
		Y++;
	X=y2dim[3]/CONV_TILE;
	if (y2dim[3]%CONV_TILE)
		X++;
  dim3 conv2_block (CONV_TILE, CONV_TILE,1);
  dim3 conv2_grid (xdims[0],X,Y);
  convLayerForwardMatrixKernel<<<conv2_grid,conv2_block>>>(conv2_input,W2,conv2_output,x2dim[1],y2dim[1],x2dim[3],y2dim[3]);
	// average pooling
  W_grid = NN_1_dim[1] / TILE_WIDTH;
  H_grid = NN_1_dim[2] / TILE_WIDTH;
  if (y1dim[1]%TILE_WIDTH){
    W_grid++;
    H_grid++;
  }
  Y = H_grid * W_grid;
  dim3 avg_pool_2_block (TILE_WIDTH, TILE_WIDTH,1);
  dim3 avg_pool_2_grid (xdims[0], NN_1_dim[3], Y);
  averagePool<<<avg_pool_2_grid,avg_pool_2_block>>>(conv2_output,NN_L1_input,W_grid,y2dim[1],NN_1_dim[1],y2dim[3]);
	dim3 fully_forward_1_grid(xdims[0],1,1);
	dim3 fully_forward_1_block(fc1dims[0],1,1);
	gpu_fully_forward<<<fully_forward_1_grid,fully_forward_1_block>>>(NN_L1_input,NN_L1_weights,NN_L2_input,fc1dims[1],fc1dims[0]);
	dim3 fully_forward_2_grid(xdims[0],1,1);
	dim3 fully_forward_2_block(fc2dims[0],1,1);
	gpu_fully_forward<<<fully_forward_2_grid,fully_forward_2_block>>>(NN_L2_input,NN_L2_weights,NN_output_gpu,fc2dims[1],fc2dims[0]);
  check_success(hipMemcpy(argmax_input,NN_output_gpu,sizeof(float)*xdims[0]*fc2dims[1],hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
  check_success(hipFree(conv1_input));
  check_success(hipFree(conv1_output));
  check_success(hipFree(conv2_input));
  check_success(hipFree(conv2_output));
  check_success(hipFree(W1));
  check_success(hipFree(W2));
  check_success(hipFree(NN_L1_input));
  check_success(hipFree(NN_L2_input));
  check_success(hipFree(NN_L1_weights));
  check_success(hipFree(NN_L2_weights));
  check_success(hipFree(NN_output_gpu));
  const int fdims[] = {xdims[0], fc2dims[1]};
  argmax(argmax_input, fdims, out);
	free(argmax_input);
}

int main(int argc, char **argv) {

  if (argc != 3 && argc != 4) {
    std::cerr << "\n"
              << "This program performs the forward opertion step for "
                 "Convolutional Neural Network(CNN).  "
                 "Sample usage: \n"
              << argv[0]
              << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
    return -1;
  }
  FLAGS_testdata = std::string(argv[1]);
  FLAGS_model    = std::string(argv[2]);
  if (argc == 3) {
    const std::map<std::string, int> default_batch_sizes{
        {"../data/test2.hdf5", 2},
        {"../data/test10.hdf5", 10},
        {"../data/test100.hdf5", 100},
        {"../data/testfull.hdf5", 10000}};
    const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
    if (batch_size_in_map == default_batch_sizes.end()) {
      std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
      return -1;
    }
    FLAGS_batch_size = batch_size_in_map->second;
  } else if (argc == 4) {
    FLAGS_batch_size = atoi(argv[3]);
  }
  xdims[0] = FLAGS_batch_size;
  rdims[0] = FLAGS_batch_size;

  // Load data into x and y
  float *x = allocate<float>(xdims);
  float *y = allocate<float>(rdims);
  loadData(x, y);

  // Load model
  float *conv1 = allocate<float>(conv1dims);
  float *conv2 = allocate<float>(conv2dims);
  float *fc1   = allocate<float>(fc1dims);
  float *fc2   = allocate<float>(fc2dims);
  loadModel(conv1, conv2, fc1, fc2);

  // Perform foward opertion
  int *out = zeros<int>(FLAGS_batch_size);

  // get start time
  const auto start = now();

  forward_operation_gpu(x, conv1, conv2, fc1, fc2, out);

  // get end time
  const auto end = now();

  // get elapsed time in milliseconds
  const auto elapsed =
      std::chrono::duration<double, std::milli>(end - start).count();

  // Get reference
  int *ref = zeros<int>(FLAGS_batch_size);
  argmax(y, rdims, ref);

  // Calculate correctness
  int num_correct = 0;
	for (const auto i : range(0, FLAGS_batch_size)) {
		if (out[i] == ref[i]) {
      num_correct++;
    }
  }
  std::cout << "Done with " << FLAGS_batch_size << " queries in "
            << "elapsed = " << elapsed << " milliseconds. Correctness: "
            << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

  delete[] x;
  delete[] y;
  delete[] conv1;
  delete[] conv2;
  delete[] fc1;
  delete[] fc2;
  delete[] out;
  delete[] ref;

  return 0;
}



